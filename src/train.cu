#include "hip/hip_runtime.h"
#include "../network.hpp"
#include "../io.hpp"
#include "../util.hpp"
#include "../composit_layer.hpp"
#include "../../picojson/picojson.h"

#include <cstdio>
#include <iostream>
#include <string>
using namespace std;

typedef unsigned char byte;



int main(){
  InitTensorEngine<gpu>();

  	picojson::value v;
  	cin >> v;
	if (std::cin.fail()) {
		std::cerr << picojson::get_last_error() << std::endl;
		return 1;
	}

	picojson::object& o = v.get<picojson::object>();
	string network_type;
	picojson::value network_param;

// data
	string data_type;
	string fn_train_data;
	string fn_train_label;
	string fn_test_data;
	string fn_test_label;
	int n_train, n_test;

// learning
	int n_time=1, n_batch=1;
	int init_epoch = 0;
	int iter_per_epoch=100;
	int snapshot_interval=100;
	int max_epoch=10000;
	bool load_model=false;
	string sgd = "rmsprop";
	float momentum = 0.95f;
	float decay = 0.001f;
	float base_lr = 0.0001f;
	float lr_mult = 0.5f;
	int lr_mult_interval = 100;
	int sampling_length = 0;
	int train_error_limit = 0;


	for (picojson::object::const_iterator i = o.begin(); i != o.end(); ++i) {
		if(i->first == "env"){
			picojson::object env = i->second.get<picojson::object>();
			for (picojson::object::const_iterator j = env.begin(); j != env.end(); ++j) {
				if(j->first == "python") Global_params::python = j->second.get<string>();
				if(j->first == "prefix") Global_params::prefix = j->second.get<string>();
				if(j->first == "debug") Global_params::DEBUG = j->second.get<bool>();
			}
		}
		if(i->first == "network"){
			picojson::object net = i->second.get<picojson::object>();
			for (picojson::object::const_iterator j = net.begin(); j != net.end(); ++j) {
				if(j->first == "type") network_type = j->second.get<string>();
				if(j->first == "param") network_param = j->second;
			}
		}
		if(i->first == "data"){
			picojson::object dat = i->second.get<picojson::object>();
			for (picojson::object::const_iterator j = dat.begin(); j != dat.end(); ++j) {
				if(j->first == "type") data_type = j->second.get<string>();
				if(j->first == "train_data") fn_train_data = j->second.get<string>();
				if(j->first == "train_label") fn_train_label = j->second.get<string>();
				if(j->first == "test_data") fn_test_data = j->second.get<string>();
				if(j->first == "test_label") fn_test_label = j->second.get<string>();
				if(j->first == "n_train") n_train = (int)j->second.get<double>();
				if(j->first == "n_test") n_test = (int)j->second.get<double>();
				if(j->first == "load_model") load_model = j->second.get<bool>();
			}
		}
		if(i->first == "learning"){
			picojson::object dat = i->second.get<picojson::object>();
			for (picojson::object::const_iterator j = dat.begin(); j != dat.end(); ++j) {

				if(j->first == "n_time") n_time = (int)j->second.get<double>();
				if(j->first == "n_batch") n_batch = (int)j->second.get<double>();
				if(j->first == "init_epoch") init_epoch = (int)j->second.get<double>();
				if(j->first == "iter_per_epoch") iter_per_epoch = (int)j->second.get<double>();
				if(j->first == "snapshot_interval") snapshot_interval = (int)j->second.get<double>();
				if(j->first == "max_epoch") max_epoch = (int)j->second.get<double>();
				if(j->first == "sgd") sgd = j->second.get<string>();
				if(j->first == "momentum") momentum = (float)j->second.get<double>();
				if(j->first == "decay") decay = (float)j->second.get<double>();
				if(j->first == "base_lr") base_lr = (float)j->second.get<double>();
				if(j->first == "lr_mult") lr_mult = (float)j->second.get<double>();
				if(j->first == "lr_mult_interval") lr_mult_interval = (int)j->second.get<double>();
				if(j->first == "sampling_length") sampling_length = (int)j->second.get<double>();
				if(j->first == "train_error_limit") train_error_limit = (int)j->second.get<double>();

			}
		}
	}

	cerr << "load json done" << endl;

	// data load
	Node<gpu> *node_train_data, *node_train_label, *node_test_data, *node_test_label;
	Node<gpu> *node_train_data0, *node_test_data0;

	if(data_type=="MNIST"){
		node_train_data = new Node<gpu>(n_train, n_batch, n_time, 784, true);
		node_train_label = new Node<gpu>(n_train, n_batch, n_time, 10, true);
		node_test_data = new Node<gpu>(n_test, n_batch, n_time, 784, true);
		node_test_label = new Node<gpu>(n_test, n_batch, n_time, 10, true);
		read_mnist_data(fn_train_data, node_train_data, n_train, 784, 16);
		read_mnist_label(fn_train_label, node_train_label, n_train, 10, 8);
		read_mnist_data(fn_test_data, node_test_data, n_test, 784, 16);
		read_mnist_label(fn_test_label, node_test_label, n_test, 10, 8);
	}else if(data_type=="w2v"){

		node_train_data = new W2vNode<gpu>(n_train, n_batch, n_time, 1001, true);
		node_train_label = new W2vNode<gpu>(n_train, n_batch, n_time, 1001, true);
		node_test_data = new W2vNode<gpu>(n_test, n_batch, n_time, 1001, true);
		node_test_label = new W2vNode<gpu>(n_test, n_batch, n_time, 1001, true);
		read_w2v_data<gpu>(fn_train_data, (W2vNode<gpu> *)node_train_data, n_train);
		read_w2v_data<gpu>(fn_train_label, (W2vNode<gpu> *)node_train_label, n_train);
		read_w2v_data<gpu>(fn_test_data, (W2vNode<gpu> *)node_test_data, n_test);
		read_w2v_data<gpu>(fn_test_label, (W2vNode<gpu> *)node_test_label, n_test);

		init_w2v<gpu>();

	}else if(data_type=="char2"){

		node_train_data = new CharacterNode2<gpu>(n_train, n_batch, n_time, 16, true, true);
		node_train_label = new CharacterNode2<gpu>(n_train, n_batch, n_time, 5000, true);
		node_test_data = new CharacterNode2<gpu>(n_test, n_batch, n_time, 16, true, true);
		node_test_label = new CharacterNode2<gpu>(n_test, n_batch, n_time, 5000, true);
		read_int_binary_data<gpu>(fn_train_data, (CharacterNode2<gpu> *)node_train_data, n_train);
		read_int_binary_data<gpu>(fn_train_label, (CharacterNode2<gpu> *)node_train_label, n_train);
		read_int_binary_data<gpu>(fn_test_data, (CharacterNode2<gpu> *)node_test_data, n_test);
		read_int_binary_data<gpu>(fn_test_label, (CharacterNode2<gpu> *)node_test_label, n_test);

		init_char2<gpu>();
	}else if(data_type=="ngram"){

		node_train_data0 = new CharacterNode2<gpu>(n_train, n_batch, n_time, 16, true, true);
		node_train_label = new CharacterNode2<gpu>(n_train, n_batch, n_time, 5000, true);
		node_test_data0 = new CharacterNode2<gpu>(n_test, n_batch, n_time, 16, true, true);
		node_test_label = new CharacterNode2<gpu>(n_test, n_batch, n_time, 5000, true);
		read_int_binary_data<gpu>(fn_train_data, (CharacterNode2<gpu> *)node_train_data0, n_train);
		read_int_binary_data<gpu>(fn_train_label, (CharacterNode2<gpu> *)node_train_label, n_train);
		read_int_binary_data<gpu>(fn_test_data, (CharacterNode2<gpu> *)node_test_data0, n_test);
		read_int_binary_data<gpu>(fn_test_label, (CharacterNode2<gpu> *)node_test_label, n_test);

		node_train_data = new NgramNode<gpu>(n_train, n_batch, n_time, 5000);
		node_test_data = new NgramNode<gpu>(n_test, n_batch, n_time, 5000);
		((NgramNode<gpu> *)node_train_data)->c_all = ((CharacterNode2<gpu> *)node_train_data0)->c_all;
		((NgramNode<gpu> *)node_test_data)->c_all = ((CharacterNode2<gpu> *)node_test_data0)->c_all;
		init_char2<gpu>();
		NgramNode<gpu>::load_ngram();
	}


	// prepare network
	Network<gpu> *network = new Network<gpu>();
	network->train_data = node_train_data;
	network->train_label = node_train_label;
	network->test_data = node_test_data;
	network->test_label = node_test_label;

	if(network_type=="MLP"){
		int layers = 0;
		int *num_neurons;
		string hidden_nl="relu", out_nl="none";
		bool dropout=false;
		string loss = "category";
		string shuffle = "random_random";

		picojson::object dat = network_param.get<picojson::object>();
		for (picojson::object::const_iterator j = dat.begin(); j != dat.end(); ++j) {
			if(j->first == "layers") layers = (int)j->second.get<double>();
			if(j->first == "neurons"){
				num_neurons = (int *)malloc(layers*sizeof(int));
				const picojson::array& a = j->second.get<picojson::array>();
				int cnt = 0;
				for(picojson::array::const_iterator i=a.begin(); i!=a.end(); ++i){
					num_neurons[cnt++] = (int)i->get<double>();
				}
			}
			if(j->first == "hidden_nl") hidden_nl = j->second.get<string>();
			if(j->first == "out_nl") out_nl = j->second.get<string>();
			if(j->first == "dropout") dropout = j->second.get<bool>();
			if(j->first == "loss") loss = j->second.get<string>();
			if(j->first == "shuffle") shuffle = j->second.get<string>();
		}
		network->loss_type = loss;
		network->shuffle_type = shuffle;
//		network->is_dropout = dropout;
		network->net = new MLP<gpu>(layers, num_neurons, out_nl, hidden_nl);
	}else if(network_type=="SLSTM"){
		int layers = 0;
		int base_neurons=512;
		int inout_dim = 1001;
		string out_nl="none";
		string loss = "mse";
		string shuffle = "w2v";

		picojson::object dat = network_param.get<picojson::object>();
		for (picojson::object::const_iterator j = dat.begin(); j != dat.end(); ++j) {
			if(j->first == "layers") layers = (int)j->second.get<double>();
			if(j->first == "base_neurons")base_neurons = (int)j->second.get<double>();
			if(j->first == "out_nl") out_nl = j->second.get<string>();
			if(j->first == "loss") loss = j->second.get<string>();
			if(j->first == "shuffle") shuffle = j->second.get<string>();
			if(j->first == "inout_dim") inout_dim = (int)j->second.get<double>();
		}
		network->loss_type = loss;
		network->shuffle_type = shuffle;
//		network->is_dropout = dropout;
		cerr << base_neurons << " " << layers << endl;
		network->net = new Stacked_LSTM<gpu>(inout_dim, base_neurons, layers, out_nl);
	}else if(network_type=="HRes"){
		int base_neurons=512;
		int inout_dim = 1001;
		string out_nl="none";
		string loss = "mse";
		string shuffle = "w2v";

		picojson::object dat = network_param.get<picojson::object>();
		for (picojson::object::const_iterator j = dat.begin(); j != dat.end(); ++j) {
			if(j->first == "base_neurons")base_neurons = (int)j->second.get<double>();
			if(j->first == "out_nl") out_nl = j->second.get<string>();
			if(j->first == "loss") loss = j->second.get<string>();
			if(j->first == "shuffle") shuffle = j->second.get<string>();
			if(j->first == "inout_dim") inout_dim = (int)j->second.get<double>();
		}
		network->loss_type = loss;
		network->shuffle_type = shuffle;
//		network->is_dropout = dropout;
//		cerr << base_neurons << " " << layers << endl;
		network->net = new Hybrid_LSTM_Reservoir<gpu>(inout_dim, base_neurons, out_nl);
	}else if(network_type=="GMR2"){
		int base_neurons=512;
		int inout_dim = 1001;
		string out_nl="none";
		string loss = "mse";
		string shuffle = "w2v";

		picojson::object dat = network_param.get<picojson::object>();
		for (picojson::object::const_iterator j = dat.begin(); j != dat.end(); ++j) {
			if(j->first == "base_neurons")base_neurons = (int)j->second.get<double>();
			if(j->first == "out_nl") out_nl = j->second.get<string>();
			if(j->first == "loss") loss = j->second.get<string>();
			if(j->first == "shuffle") shuffle = j->second.get<string>();
			if(j->first == "inout_dim") inout_dim = (int)j->second.get<double>();
		}
		network->loss_type = loss;
		network->shuffle_type = shuffle;
//		network->is_dropout = dropout;
//		cerr << base_neurons << " " << layers << endl;
		network->net = new Gate_MLP_Reservoir2<gpu>(inout_dim, base_neurons, out_nl);
	}else if(network_type=="GMR2S"){
		int base_neurons=512;
		int inout_dim = 1001;
		string out_nl="none";
		string loss = "mse";
		string shuffle = "w2v";

		picojson::object dat = network_param.get<picojson::object>();
		for (picojson::object::const_iterator j = dat.begin(); j != dat.end(); ++j) {
			if(j->first == "base_neurons")base_neurons = (int)j->second.get<double>();
			if(j->first == "out_nl") out_nl = j->second.get<string>();
			if(j->first == "loss") loss = j->second.get<string>();
			if(j->first == "shuffle") shuffle = j->second.get<string>();
			if(j->first == "inout_dim") inout_dim = (int)j->second.get<double>();
		}
		network->loss_type = loss;
		network->shuffle_type = shuffle;
//		network->is_dropout = dropout;
//		cerr << base_neurons << " " << layers << endl;
		network->net = new Gate_MLP_Reservoir2_SLSTM<gpu>(inout_dim, base_neurons, out_nl);
	}else if(network_type=="Hatsuwa_simple"){
		int base_neurons=512;
		int inout_dim = 1001;
		string out_nl="none";
		string loss = "mse";
		string shuffle = "w2v";

		picojson::object dat = network_param.get<picojson::object>();
		for (picojson::object::const_iterator j = dat.begin(); j != dat.end(); ++j) {
			if(j->first == "base_neurons")base_neurons = (int)j->second.get<double>();
			if(j->first == "out_nl") out_nl = j->second.get<string>();
			if(j->first == "loss") loss = j->second.get<string>();
			if(j->first == "shuffle") shuffle = j->second.get<string>();
			if(j->first == "inout_dim") inout_dim = (int)j->second.get<double>();
		}
		network->loss_type = loss;
		network->shuffle_type = shuffle;
//		network->is_dropout = dropout;
//		cerr << base_neurons << " " << layers << endl;
		network->net = new Hatsuwa_simple<gpu>();
	}else if(network_type=="Hatsuwa_ngram"){
		int base_neurons=512;
		int inout_dim = 1001;
		string out_nl="none";
		string loss = "mse";
		string shuffle = "w2v";

		picojson::object dat = network_param.get<picojson::object>();
		for (picojson::object::const_iterator j = dat.begin(); j != dat.end(); ++j) {
			if(j->first == "base_neurons")base_neurons = (int)j->second.get<double>();
			if(j->first == "out_nl") out_nl = j->second.get<string>();
			if(j->first == "loss") loss = j->second.get<string>();
			if(j->first == "shuffle") shuffle = j->second.get<string>();
			if(j->first == "inout_dim") inout_dim = (int)j->second.get<double>();
		}
		network->loss_type = loss;
		network->shuffle_type = shuffle;
//		network->is_dropout = dropout;
//		cerr << base_neurons << " " << layers << endl;
		network->net = new Hatsuwa_ngram<gpu>();
	}else if(network_type=="Hatsuwa_aws_res"){
		int base_neurons=512;
		int inout_dim = 1001;
		int layers = 0;
		string out_nl="none";
		string loss = "mse";
		string shuffle = "w2v";

		picojson::object dat = network_param.get<picojson::object>();
		for (picojson::object::const_iterator j = dat.begin(); j != dat.end(); ++j) {
			if(j->first == "base_neurons")base_neurons = (int)j->second.get<double>();
			if(j->first == "out_nl") out_nl = j->second.get<string>();
			if(j->first == "loss") loss = j->second.get<string>();
			if(j->first == "shuffle") shuffle = j->second.get<string>();
			if(j->first == "inout_dim") inout_dim = (int)j->second.get<double>();
			if(j->first == "layers") layers = (int)j->second.get<double>();
		}
		network->loss_type = loss;
		network->shuffle_type = shuffle;
//		network->is_dropout = dropout;
//		cerr << base_neurons << " " << layers << endl;
		network->net = new Hatsuwa_aws_res<gpu>(base_neurons, base_neurons, layers);
	}else if(network_type=="Hatsuwa_aws_nores"){
		int base_neurons=512;
		int inout_dim = 1001;
		int layers = 0;
		string out_nl="none";
		string loss = "mse";
		string shuffle = "w2v";

		picojson::object dat = network_param.get<picojson::object>();
		for (picojson::object::const_iterator j = dat.begin(); j != dat.end(); ++j) {
			if(j->first == "base_neurons")base_neurons = (int)j->second.get<double>();
			if(j->first == "out_nl") out_nl = j->second.get<string>();
			if(j->first == "loss") loss = j->second.get<string>();
			if(j->first == "shuffle") shuffle = j->second.get<string>();
			if(j->first == "inout_dim") inout_dim = (int)j->second.get<double>();
			if(j->first == "layers") layers = (int)j->second.get<double>();
		}
		network->loss_type = loss;
		network->shuffle_type = shuffle;
//		network->is_dropout = dropout;
//		cerr << base_neurons << " " << layers << endl;
		network->net = new Hatsuwa_aws_nores<gpu>(base_neurons, base_neurons, layers);
	}
	network->net->set_param("eta", base_lr);
	network->net->set_param("decay", decay);
	network->net->set_param("sgd_algo", sgd=="momentum"?0:1);
	network->net->set_param("momentum", momentum);

	cerr << "network initialize done" << endl;

	// learning

	if(load_model){
		FILE *ii = fopen(to_string("./tmp/"+Global_params::prefix+"_model", init_epoch).c_str(), "rb");
		network->load_model(ii);
		fclose(ii);
	}

	for(int epoch=init_epoch+1; epoch<=max_epoch; epoch++){
		network->train(1, iter_per_epoch);

		float train_err = network->train_error(train_error_limit);
		float test_err = network->test_error();
		cerr << "epoch: " << epoch << ", train error: " << train_err << ", test error: " << test_err << endl;
		printf("epoch: %d, train error: %f, test error: %f\n", epoch, train_err, test_err);
		if(epoch%lr_mult_interval==0){
			float eta = network->net->get_param("eta");
	        network->net->set_param("eta", eta*lr_mult);
			cerr << "eta: " << eta << endl;
		}
		if(epoch%snapshot_interval==0){
			if(sampling_length>0) network->sampling_char2(to_string("./tmp/"+Global_params::prefix+"_smp", epoch), sampling_length);
			FILE *ir = fopen(to_string("./tmp/"+Global_params::prefix+"_model", epoch).c_str(), "wb");
			network->save_model(ir);
			fclose(ir);
		}

	}

  ShutdownTensorEngine<gpu>();
	return 0;
}
